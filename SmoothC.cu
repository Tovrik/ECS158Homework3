#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <string.h>
using namespace std;

__global__ void smoothc(float *x, float *y, float *m, int n, float h) {
	/*
	blockDim.x => gives the number of threads in a block, in the particular direction
	gridDim.x => gives the number of blocks in a grid
	*/

  int blockIndex = (blockIdx.x * blockDim.x) + threadIdx.x;

  //printf("threadIdx.x = %d\n", threadIdx.x);
  //printf("blockDim.x = %d\n", blockDim.x);

  float sum = 0;
  int count = 0;

  for(int i = 0; i < n; i++) {
    if(fabsf(x[blockIndex] - x[i]) < h) {
       //printf("x[blockIndex] = %d and x[i] = %d\n", x[blockIndex], x[i]);
       sum = sum + y[i];
       count = count + 1;
    }
  }
  //__syncthreads(); 
  //printf("sum = %f\n", sum);
  //printf("count = %d\n", count);
  m[blockIndex] = sum / count;
}

int main(int argc, char** argv) {
  hipDeviceProp_t Props ;
  hipGetDeviceProperties(&Props , 0) ;
  // Declare and allocate host and device memory

  // Host memory arrays
  int n = 2500;
  float h = 2;
  float x[n];
  float y[n];
  float averageArrays[n];   
  memset(averageArrays, 0, sizeof(averageArrays));
  for(int i = 1000, j = n; i < n+1000; i++, j++) {
    x[i-1000] = i + 1;
    y[i-1000] = j + 1;
  }
  // printf("x[0] = %f\n", x[0]);
  // printf("x[2499] = %f\n", x[2499]);
  // printf("y[0] = %f\n", y[0]);
  // printf("y[2499] = %f\n", y[2499]);

  // Device Memory pointers
  float *xchunk;
  float *ychunk;
  float *avgsPointer;

  // Allocate memory on the device
  hipMalloc((void**) &xchunk, sizeof(float) * n);
  hipMalloc((void**) &ychunk, sizeof(float) * n);
  hipMalloc((void**) &avgsPointer, sizeof(float) * n);

  // Transfer the host arrays to Device
  hipMemcpy(xchunk, x, sizeof(float) * n, hipMemcpyHostToDevice);
  hipMemcpy(ychunk, y, sizeof(float) * n, hipMemcpyHostToDevice);
  hipMemcpy(avgsPointer, averageArrays, sizeof(float)* n, hipMemcpyHostToDevice);

  // Set up Parameters for threads structure
  // dim3 dimGrid(n, 1);
  // dim3 dimBlock(1, 1, 1);

  int totalBlocks = 0;
  int threads_per_block = 0;

  if(n < (Props.maxThreadsPerBlock-1) ){
  	totalBlocks = 1;
  	threads_per_block = n;
  }
  else {
  	totalBlocks = (int)(ceil((float)n / (Props.maxThreadsPerBlock/2) ));
  	threads_per_block = Props.maxThreadsPerBlock / 2;
  }
  printf("MAX THREADS CUDA = %d\n", Props.maxThreadsPerBlock);
  printf("total blocks = %d\n", totalBlocks);
  printf("threads/block = %d\n", threads_per_block);

  // Invoke the kernel
  smoothc <<<totalBlocks, threads_per_block>>> (xchunk, ychunk, avgsPointer, n, h);
  // Wait for kernel to finish()
  hipDeviceSynchronize();
  // Copy from device to host. 
  hipMemcpy(averageArrays, avgsPointer, sizeof(float)*n, hipMemcpyDeviceToHost);

  for(int i = 0; i < n; i++) {
    cout << averageArrays[i] << endl;
  }
  // Free memory
  hipFree(xchunk);
  hipFree(ychunk);
  hipFree(avgsPointer);
  return 0;
}