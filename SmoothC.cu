#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <string.h>
using namespace std;

__global__ void smoothc(float *x, float *y, float *m, int n, float h) {
	/*
	blockDim.x => gives the number of threads in a block, in the particular direction
	gridDim.x => gives the number of blocks in a grid
	*/

  int blockIndex = (blockIdx.x * blockDim.x) + threadIdx.x;

  printf("threadIdx.x = %d\n", threadIdx.x);
  printf("blockDim.x = %d\n", blockDim.x);

  float sum = 0;
  int count = 0;

  for(int i = 0; i < blockDim.x*gridDim.x; i++) {
    if(fabsf(x[blockIndex] - x[i]) < h) {
       //printf("x[blockIndex] = %d and x[i] = %d\n", x[blockIndex], x[i]);
       sum = sum + y[i];
       count = count + 1;
    }
  }
  //__syncthreads(); 
  printf("sum = %f\n", sum);
  printf("count = %d\n", count);
  m[blockIndex] = sum / count;
}

int main(int argc, char** argv) {
  hipDeviceProp_t Props ;
  hipGetDeviceProperties(&Props , 0) ;
  // Declare and allocate host and device memory

  // Host memory arrays
  int n = 5000;
  float h = 2;
  float x[n];
  float y[n];
  float averageArrays[n];   
  memset(averageArrays, 0, sizeof(averageArrays));
  for(int i = 0, j = n; i < n; i++, j++) {
    x[i] = i + 1;
    y[i] = j + 1;
  }

  // Device Memory pointers
  float *xchunk;
  float *ychunk;
  float *avgsPointer;

  // Allocate memory on the device
  hipMalloc((void**) &xchunk, sizeof(float) * n);
  hipMalloc((void**) &ychunk, sizeof(float) * n);
  hipMalloc((void**) &avgsPointer, sizeof(float) * n);

  // Transfer the host arrays to Device
  hipMemcpy(xchunk, x, sizeof(float) * n, hipMemcpyHostToDevice);
  hipMemcpy(ychunk, y, sizeof(float) * n, hipMemcpyHostToDevice);
  hipMemcpy(avgsPointer, averageArrays, sizeof(float)* n, hipMemcpyHostToDevice);

  // Set up Parameters for threads structure
  // dim3 dimGrid(n, 1);
  // dim3 dimBlock(1, 1, 1);

  int totalBlocks = 0;
  int threads_per_block = 0;

  if(n < (Props.maxThreadsPerBlock-1) ){
  	totalBlocks = 1;
  	threads_per_block = n;
  }
  else {
  	totalBlocks = (int)(ceil((float)n / (Props.maxThreadsPerBlock/2) ));
  	threads_per_block = Props.maxThreadsPerBlock / 2;
  }
  printf("MAX THREADS CUDA = %d\n", Props.maxThreadsPerBlock);
  printf("total blocks = %d\n", totalBlocks);
  printf("threads/block = %d\n", threads_per_block);

  // Invoke the kernel
  smoothc <<<totalBlocks, threads_per_block>>> (xchunk, ychunk, avgsPointer, n, h);
  // Wait for kernel to finish()
  hipDeviceSynchronize();
  // Copy from device to host. 
  hipMemcpy(averageArrays, avgsPointer, sizeof(float)*n, hipMemcpyDeviceToHost);

  for(int i = 0; i < n; i++) {
    cout << averageArrays[i] << endl;
  }
  // Free memory
  hipFree(xchunk);
  hipFree(ychunk);
  hipFree(avgsPointer);
  return 0;
}