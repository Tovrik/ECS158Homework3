#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <string.h>
using namespace std;

__global__ void smoothc(float *x, float *y, float *m, int n, float h) {
  int blockIndex = threadIdx.x;
  //printf("blockDim.x = %d\n", blockDim.x);
  float sum = 0;
  int count = 0;

  for(int i = 0; i < blockDim.x; i++) {
    if(fabsf(x[blockIndex] - x[i]) < h) {
       //printf("x[blockIndex] = %d and x[i] = %d\n", x[blockIndex], x[i]);
       sum = sum + y[i];
       count = count + 1;
    }
  }
  // __syncthreads();  
  m[blockIndex] = sum / count;
}

int main(int argc, char** argv) {
  // Declare and allocate host and device memory

  // Host memory arrays
  int n = 10;
  float h = 2;
  float x[n];
  float y[n];
  float averageArrays[n];   
  memset(averageArrays, 0, sizeof(averageArrays));
  for(int i = 0, j = n; i < n; i++, j++) {
    x[i] = i + 1;
    y[i] = j + 1;
  }

  // Device Memory pointers
  float *xchunk;
  float *ychunk;
  float *avgsPointer;

  // Allocate memory on the device
  hipMalloc((void**) &xchunk, sizeof(float) * 100);
  hipMalloc((void**) &ychunk, sizeof(float) * 100);
  hipMalloc((void**) &avgsPointer, sizeof(float) * 100);

  // Transfer the host arrays to Device
  hipMemcpy(xchunk, x, sizeof(float) * n, hipMemcpyHostToDevice);
  hipMemcpy(ychunk, y, sizeof(float) * n, hipMemcpyHostToDevice);
  hipMemcpy(avgsPointer, averageArrays, sizeof(float)* n, hipMemcpyHostToDevice);

  // Set up Parameters for threads structure
  dim3 dimGrid(n, 1);
  dim3 dimBlock(1, 1, 1);

  // Invoke the kernel
  smoothc <<<1, n>>> (xchunk, ychunk, avgsPointer, n, h);
  // Wait for kernel to finish()
  hipDeviceSynchronize();
  // Copy from device to host. 
  hipMemcpy(averageArrays, avgsPointer, sizeof(float)*n, hipMemcpyDeviceToHost);

  for(int i = 0; i < n; i++) {
    cout << averageArrays[i] << endl;
  }
  // Free memory
  hipFree(xchunk);
  hipFree(ychunk);
  hipFree(avgsPointer);
  return 0;
}